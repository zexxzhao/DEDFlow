#include "hip/hip_runtime.h"
#include <type_traits>
#include <hipblas.h>

#include "alloc.h"
#include "indexing.h"
#include "Mesh.h"
#include "Field.h"
#include "Array.h"
#include "csr.h"

#include "assemble.h"

#define kRHOC (0.5)
#define kDT (0.1)
#define kALPHAM ((3.0 - kRHOC) / (1.0 + kRHOC))
#define kALPHAF (1.0 / (1.0 + kRHOC))
#define kGAMMA (0.5 + kALPHAM - kALPHAF)

#define NQR (4)

__constant__ f64 gw[4] = {0.0416666666666667, 0.0416666666666667, 0.0416666666666667, 0.0416666666666667};
__constant__ f64 shlu[16] = {0.5854101966249685, 0.1381966011250105, 0.1381966011250105, 0.1381966011250105,
														 0.1381966011250105, 0.5854101966249685, 0.1381966011250105, 0.1381966011250105,
														 0.1381966011250105, 0.1381966011250105, 0.5854101966249685, 0.1381966011250105,
														 0.1381966011250105, 0.1381966011250105, 0.1381966011250105, 0.5854101966249685};

__constant__ f64 shlgradu[12] = {-1.0, -1.0, -1.0,
																 1.0, 0.0, 0.0,
																 0.0, 1.0, 0.0,
																 0.0, 0.0, 1.0};


template<typename I, typename T, I bs=1, I NSHL=4>
__global__ void LoadElementValueKernel(I n, const I* ien, const I* index, const T* src, T* dst) {
	I idx = blockIdx.x * blockDim.x + threadIdx.x;
	I num_thread = blockDim.x * gridDim.x;
	while(idx < n) {
		#pragma unroll
		for(I i = 0; i < NSHL; ++i) {
			I node_id = ien[index[idx] * NSHL + i];
			#pragma unroll
			for(I j = 0; j < bs; ++j) {
				dst[idx * bs + j] = src[node_id * bs + j];
			}
		}
		idx += num_thread;
	}
}


template<typename I, typename T, I bs=1, I NSHL=4>
__global__ void LoadElementValueAXPYKernel(I n, const I* ien, const I* index, const T* src, T* dst, T alpha) {
	I idx = blockIdx.x * blockDim.x + threadIdx.x;
	I num_thread = blockDim.x * gridDim.x;
	while(idx < n) {
		#pragma unroll
		for(I i = 0; i < NSHL; ++i) {
			I node_id = ien[index[idx] * NSHL + i];
			#pragma unroll
			for(I j = 0; j < bs; ++j) {
				dst[idx * bs + j] = fma(src[node_id * bs + j], alpha, dst[idx * bs + j]);
			}
		}
		idx += num_thread;
	}
}


template <typename I, typename T, I bs=1, I NSHL=4> __global__ void
ElemRHSLocal2GlobalKernel(I num_elem, const I* iel, const I* ien, const T* elem_F, T* F) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	for(int j = 0; j < num_elem * NSHL; j += blockDim.x * gridDim.x) {
		int vertex_id = ien[iel[j] * NSHL + j % NSHL];
		#pragma unroll
		for(int j = 0; j < bs; ++j) {
			F[vertex_id * bs + j] += elem_F[j * bs + j];
		}
	}
}


template<typename I, typename T, I NSHL=4>
__global__ void GetElemJ3DKernel(I ne, const I* ien, const I* index, const T* xg, T* elemJ) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	I elem_id = index[idx];
	const T* xg_ptr0 = xg + ien[elem_id*NSHL+0] * 3;
	const T* xg_ptr1 = xg + ien[elem_id*NSHL+1] * 3;
	const T* xg_ptr2 = xg + ien[elem_id*NSHL+2] * 3;
	const T* xg_ptr3 = xg + ien[elem_id*NSHL+3] * 3;
	
	T* elemJ_ptr = elemJ + idx * 9;
	elemJ_ptr[0] = xg_ptr1[0] - xg_ptr0[0];
	elemJ_ptr[1] = xg_ptr1[1] - xg_ptr0[1];
	elemJ_ptr[2] = xg_ptr1[2] - xg_ptr0[2];
	elemJ_ptr[3] = xg_ptr2[0] - xg_ptr0[0];
	elemJ_ptr[4] = xg_ptr2[1] - xg_ptr0[1];
	elemJ_ptr[5] = xg_ptr2[2] - xg_ptr0[2];
	elemJ_ptr[6] = xg_ptr3[0] - xg_ptr0[0];
	elemJ_ptr[7] = xg_ptr3[1] - xg_ptr0[1];
	elemJ_ptr[8] = xg_ptr3[2] - xg_ptr0[2];

}
template<typename I, typename T, I NSHL=4>
__global__ void GetElemDetJKernel(I ne, T* elem_metric) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >= ne) return;

	T* elem_lu = elem_metric + idx * 10; /* elem_lu is the LU decomposition of the Jacobian matrix */
	T* elem_detJ = elem_lu + 9; /* elem_detJ is the determinant of the Jacobian matrix */
	*elem_detJ = elem_lu[0] * elem_lu[4] * elem_lu[8];
}

template<typename I, typename T, I NSHL=4>
__global__ void CopyElemJacobiansKernel(I ne, const T** elemJ, T* elemJ_ptr) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >= ne) return;

	const T* elemJ_ptr0 = elemJ[idx];
	T* elemJ_ptr1 = elemJ_ptr + idx * 10;
	elemJ_ptr1[0] = elemJ_ptr0[0];
	elemJ_ptr1[1] = elemJ_ptr0[1];
	elemJ_ptr1[2] = elemJ_ptr0[2];
	elemJ_ptr1[3] = elemJ_ptr0[3];
	elemJ_ptr1[4] = elemJ_ptr0[4];
	elemJ_ptr1[5] = elemJ_ptr0[5];
	elemJ_ptr1[6] = elemJ_ptr0[6];
	elemJ_ptr1[7] = elemJ_ptr0[7];
	elemJ_ptr1[8] = elemJ_ptr0[8];
}

/**
 * @brief Calculate the inverse of the Jacobian matrix of the element
 * @param[in] ne   The number of elements
 * @param[in] ien  The index of the element in the mesh
 * @param[in] xg   The global coordinates of the nodes
 * @param[out] elem_metric The inverse of the Jacobian matrix of the element
 */
template<typename I, typename T, I NSHL=4>
void GetElemInvJ3DKernel(I ne, const I* ien, const I* index, const T* xg, T* elem_metric) {
	int block_size = 1024;
	int num_block = (ne + block_size - 1) / block_size;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	GetElemJ3DKernel<<<num_block, block_size>>>(ne, ien, index, xg, elem_metric);

	const int BATCH_SIZE = 128;
	int* info = (int*)CdamMallocDevice(BATCH_SIZE * sizeof(int));
	int* pivot = (int*)CdamMallocDevice(BATCH_SIZE * 3 * sizeof(int));

	T* elem_metric_batch[BATCH_SIZE];
	T* elem_batch[BATCH_SIZE];

	T** d_elem_metric_batch = (T**)CdamMallocDevice(BATCH_SIZE * sizeof(T*));
	T** d_elem_batch = (T**)CdamMallocDevice(BATCH_SIZE * sizeof(T*));


	T* elem_batch_buff = (T*)CdamMallocDevice(BATCH_SIZE * 10 * sizeof(T));
	for(int j = 0; j < BATCH_SIZE; ++j) {
		elem_batch[j] = elem_batch_buff + j * 10;
	}
	hipMemcpy(d_elem_batch, elem_batch, BATCH_SIZE * sizeof(T*), hipMemcpyHostToDevice);


	for(int i = 0; i < ne; i += BATCH_SIZE) {
		int num_elem = (i + BATCH_SIZE < ne) ? BATCH_SIZE : ne - i;
		for(int j = 0; j < num_elem; ++j) {
			elem_metric_batch[j] = elem_metric+ (i + j) * 10;
		}

		hipMemcpy(d_elem_metric_batch, elem_metric_batch, BATCH_SIZE * sizeof(T*), hipMemcpyHostToDevice);

		if constexpr (std::is_same_v<T, double>) {
			hipblasDgetrfBatched(handle, 3, elem_metric_batch, 3, pivot, info, num_elem);
			GetElemDetJKernel<<<num_block, block_size>>>(num_elem, elem_metric + i * 10);
			hipblasDgetriBatched(handle, 3, elem_metric_batch, 3, pivot, elem_batch, 3, info, num_elem);
		}
		else if constexpr (std::is_same_v<T, float>) {
		}
		else if constexpr (std::is_same_v<T, hipDoubleComplex>) {
		}
		else if constexpr (std::is_same_v<T, hipFloatComplex>) {
		}

		CopyElemJacobiansKernel<I, T><<<num_block, block_size>>>((I)BATCH_SIZE, (const T**)elem_batch, elem_metric + i * 10);
	}

	CdamFreeDevice(info, BATCH_SIZE * sizeof(int));
	CdamFreeDevice(pivot, BATCH_SIZE * 3 * sizeof(int));
	CdamFreeDevice(elem_batch_buff, BATCH_SIZE * 10 * sizeof(T));
	CdamFreeDevice(d_elem_metric_batch, BATCH_SIZE * sizeof(T*));
	CdamFreeDevice(d_elem_batch, BATCH_SIZE * sizeof(T*));
}



__BEGIN_DECLS__

/**
* @brief Load the values from the source array to the destination array
* @param[in]  count The number of values to be loaded
* @param[in]  index The index of the values to be loaded, whose size is (count*inc,)
* @param[in]  inc   The increment of the index
* @param[in]  bs    The length of the block
* @param[in]  src   The source array to be loaded. The array is column-majored of size (block_length, *)
* @param[out] dst   The destination array to be loaded. The array is column-majored of size (block_length, count)
*/
static __global__ void
LoadValueKernel(u32 count, const u32* index, u32 inc, u32 block_length, const f64* src, f64* dst) {

	i32 idx = blockIdx.x * blockDim.x + threadIdx.x; 
	if(idx >= count) return;

	const u32* index_ptr = index + idx * inc;
	f64* dst_ptr = dst + idx * block_length;
	while(block_length >= 4) {
		*((double4*)dst_ptr) = *((double4*)(src + index_ptr[0]));
		block_length -= 4;
		dst_ptr += 4;
		index_ptr += 4;
	}
	if(block_length >= 2) {
		*((double2*)dst_ptr) = *((double2*)(src + index_ptr[0]));
		block_length -= 2;
		dst_ptr += 2;
		index_ptr += 2;
	}
	if(block_length >= 1) {
		*dst_ptr = src[index_ptr[0]];
	}

}

/**
 * @brief Load the values from the source array and apply axpy operation to the destination array
 */
static __global__ void
LoadValueAxpyKernel(u32 count, const u32* index, u32 inc, u32 block_length, const f64* src, f64* dst, f64 alpha) {
	i32 idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >= count) return;

	const u32* index_ptr = index + idx * inc;
	f64* dst_ptr = dst + idx * block_length;
	while(block_length >= 4) {
		// *((double4*)dst_ptr) = fma(*((double4*)(src + index_ptr[0])), alpha, *((double4*)dst_ptr));
		dst_ptr[0] += alpha * src[index_ptr[0]];
		dst_ptr[1] += alpha * src[index_ptr[1]];
		dst_ptr[2] += alpha * src[index_ptr[2]];
		dst_ptr[3] += alpha * src[index_ptr[3]];
		block_length -= 4;
		dst_ptr += 4;
		index_ptr += 4;
	}
	if(block_length >= 2) {
		dst_ptr[0] += alpha * src[index_ptr[0]];
		dst_ptr[1] += alpha * src[index_ptr[1]];
		block_length -= 2;
		dst_ptr += 2;
		index_ptr += 2;
	}
	if(block_length >= 1) {
		*dst_ptr += alpha * src[index_ptr[0]];
	}
}

static __global__ void
GetShapeGradKernel(u32 num_elem, const f64* elem_invJ, f64* shgradl) {
	i32 idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >= num_elem) return;

	const f64* elem_invJ_ptr = elem_invJ + idx * 10;
	f64* shgradl_ptr = shgradl + idx * 12;
	f64 dxidx[9] = {elem_invJ_ptr[0], elem_invJ_ptr[1], elem_invJ_ptr[2],
									elem_invJ_ptr[3], elem_invJ_ptr[4], elem_invJ_ptr[5],
									elem_invJ_ptr[6], elem_invJ_ptr[7], elem_invJ_ptr[8]};

	#pragma unroll
	for(u32 s = 0; s < 4; ++s) {
		f64 shgradu[3] = {shlgradu[s*3+0], shlgradu[s*3+1], shlgradu[s*3+2]};
		shgradl_ptr[s*3+0] = shgradu[0] * dxidx[0] + shgradu[1] * dxidx[3] + shgradu[2] * dxidx[6];
		shgradl_ptr[s*3+1] = shgradu[0] * dxidx[1] + shgradu[1] * dxidx[4] + shgradu[2] * dxidx[7];
		shgradl_ptr[s*3+2] = shgradu[0] * dxidx[2] + shgradu[1] * dxidx[5] + shgradu[2] * dxidx[8];
	}
}
static __global__ void
AssemleTetWeakFormKernel(u32 num_elem, const u32* iel, const f64* elem_invJ,
												 const f64* qr_wgalpha, const f64* qr_dwgalpha, const f64* qr_wggradalpha,
												 const f64* shgradl,
												 f64* elem_F, f64* elem_J) {
	const int NSHL = 4;
	i32 idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx >= num_elem) return;

	const f64* elem_invJ_ptr = elem_invJ + idx * 10;
	const f64 detJ = elem_invJ_ptr[9];
	const f64* qr_wgalpha_ptr = qr_wgalpha + idx * 4;
	const f64* qr_dwgalpha_ptr = qr_dwgalpha + idx * 4;
	const f64* qr_wggradalpha_ptr = qr_wggradalpha + idx * 3;
	const f64* shgradl_ptr = shgradl + idx * 12;

	f64* elem_F_ptr = elem_F + idx * 4;
	f64* elem_J_ptr = elem_J + idx * 16;

	f64 F_ptr[4+16] = {0.0, 0.0, 0.0, 0.0,
								 0.0, 0.0, 0.0, 0.0,
								 0.0, 0.0, 0.0, 0.0,
								 0.0, 0.0, 0.0, 0.0,
								 0.0, 0.0, 0.0, 0.0};
	f64* J_ptr = F_ptr + 4;
	
	/* Weak form: */
	/* F_A[0:NSHL] += (gw[0:NQR] * detJ * qr_dwgalpha_ptr[0:NQR]) @ shl[0:NQR, 0:NSHL] */
	/* F_A[0:NSHL] += sum(gw[0:NQR]) * detJ * qr_wggradalpha_ptr[0:3] @ shgradl[0:3, 0:NSHL] */
	/* J_AB[0:NSHL, 0:NSHL] += kALPHAM * detJ * shl[0,NQR, NSHL].T @ (gw[0:NQR, None] * shl[0:NQR, 0:NSHL]) */
	/* J_AB[0:NSHL, 0:NSHL] += kDT * kALPHAF * kGAMMA * sum(gw[0:NQR]) * detJ 
													 * shgradl[0:3, 0:NSHL].T @ shgradl[0:3, 0:NSHL] */

	if(elem_F) {
		#pragma unroll
		for(u32 q = 0; q < NQR; ++q) {
			f64 qr_dwgalpha_q = qr_dwgalpha_ptr[q];
			f64 qr_wggradalpha_q = qr_wggradalpha_ptr[q];
			#pragma unroll
			for(u32 s = 0; s < NSHL; ++s) {
				F_ptr[s] += gw[q] * detJ * qr_dwgalpha_q * shgradl_ptr[q*3+s];
				F_ptr[s] += gw[q] * detJ * qr_wggradalpha_q * shgradl_ptr[q*3+s];
			}
		}

		#pragma unroll
		for(u32 s = 0; s < NSHL; ++s) {
			#pragma unroll
			for(u32 t = 0; t < NSHL; ++t) {
				elem_F_ptr[s] += F_ptr[s*NSHL+t];
			}
		}
	}
	if(elem_J) {
		#pragma unroll
		for(u32 q = 0; q < NQR; ++q) {
			#pragma unroll
			for(u32 s = 0; s < NSHL; ++s) {
				#pragma unroll
				for(u32 t = 0; t < NSHL; ++t) {
					J_ptr[s*NSHL+t] += kALPHAM * detJ * shlgradu[q*3+s] * gw[q] * shlgradu[q*3+t];
					J_ptr[s*NSHL+t] += kDT * kALPHAF * kGAMMA * detJ * shgradl_ptr[q*3+s] * shgradl_ptr[q*3+t];
				}
			}
		}

		#pragma unroll
		for(u32 s = 0; s < NSHL; ++s) {
			#pragma unroll
			for(u32 t = 0; t < NSHL; ++t) {
				elem_J_ptr[s*NSHL+t] = J_ptr[s*NSHL+t];
			}
		}
	
	}
}

/**
 * @brief Assemble RHS for the tetrahedral elements
 * @param[in] ne The number of elements
 * @param[in] elem_metric The metric of the elements. The metric is column-majored of size (10, ne), where the first 9 elements are the inverse of the Jacobian matrix and the last element is the determinant of the Jacobian matrix
 * @param[in] shgradg The gradient of the shape functions. The gradient is column-majored of size (12, ne)
 * @param[in] qr_dwgalpha The quadrature weights of the gradient of the field. The array is column-majored of size (NQR, ne)
 * @param[in] qr_wgalpha The quadrature weights of the field. The array is column-majored of size (NQR, ne)
 * @param[in] qr_wggradalpha The quadrature weights of the gradient of the field. The array is column-majored of size (3, ne)
 * @param[out] elem_F The elementwise residual vector. The array is column-majored of size (NSHL, ne)
 */
void IntElemAssVec(u32 ne,
									 const f64* elem_metric, const f64* shgradg,
									 const f64* qr_dwgalpha, const f64* qr_wgalpha, const f64* qr_wggradalpha,
									 f64* elem_F) {
	/* 0. elem_F[0:NSHL, :] = 0.0 */
	hipMemset(elem_F, 0, ne * sizeof(f64) * 4);
	/* 1. elem_F[0:NSHL, :] += \sum_{q=0}^{NQR-1} qr_dwgalpha[q, :] * gw[q] * detJ * shlu[q, 0:NSHL] */
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	const f64 one = 1.0, zero = 0.0;
	for(u32 q = 0; q < NQR; ++q) {
		hipblasDger(handle, 4, ne, &one, qr_dwgalpha + q, NQR, shgradg + q * 4, 1, elem_F, 4);
	}
	

	hipblasDestroy(handle);
}


void AssembleSystemTet(Mesh3D *mesh,
											 Field *wgold, Field* dwgold, Field* dwg,
											 f64* F, Matrix* J) {

	const i32 NSHL = 4;
	const i32 MAX_BATCH_SIZE = 1 << 10;
	const u32 num_tet = Mesh3DNumNode(mesh);
	const Mesh3DData* dev = Mesh3DDevice(mesh);
	const u32* ien = Mesh3DDataTet(dev);
	const f64* xg = Mesh3DDataCoord(dev);

	f64* wgold_dptr = ArrayData(FieldDevice(wgold));
	f64* dwgold_dptr = ArrayData(FieldDevice(dwgold));
	f64* dwg_dptr = ArrayData(FieldDevice(dwg));


	u32* batch_index_ptr = NULL;
	u32 max_batch_size = 0, batch_size = 0;
	hipblasHandle_t handle;
	const f64 one = 1.0, zero = 0.0, minus_one = -1.0;

	hipblasCreate(&handle);
	f64* A_batch[MAX_BATCH_SIZE];
	f64* B_batch[MAX_BATCH_SIZE];
	f64* C_batch[MAX_BATCH_SIZE];

	for(u32 b = 0; b < mesh->num_color; ++b) {
		batch_size = CountValueColor(mesh->color, num_tet, b);
		if (batch_size > max_batch_size) {
			max_batch_size = batch_size;
		}
	}
	batch_index_ptr = (u32*)CdamMallocDevice(max_batch_size * sizeof(u32));

	f64* buffer = (f64*)CdamMallocDevice(max_batch_size * sizeof(f64));
	f64* elem_invJ = (f64*)CdamMallocDevice(max_batch_size * sizeof(f64) * 10);
	f64* shgradg = (f64*)CdamMallocDevice(max_batch_size * sizeof(f64) * NSHL * 3);

	f64* elem_F = (f64*)CdamMallocDevice(max_batch_size * sizeof(f64) * NSHL);
	f64* elem_J = (f64*)CdamMallocDevice(max_batch_size * sizeof(f64) * NSHL * NSHL);

	f64* qr_wgalpha = (f64*)CdamMallocDevice(max_batch_size * sizeof(f64) * NQR);
	f64* qr_dwgalpha = (f64*)CdamMallocDevice(max_batch_size * sizeof(f64) * NQR);
	f64* qr_wggradalpha = (f64*)CdamMallocDevice(max_batch_size * sizeof(f64) * 3);


	/* Assume all 2D arraies are column-majored */
	for(u32 b = 0; b < mesh->num_color; ++b) {

		i32 num_thread = 256;
		i32 num_block = (batch_size + num_thread - 1) / num_thread;
	
		batch_size = CountValueColor(mesh->color, num_tet, b);
		FindValueColor(mesh->color, num_tet, b, batch_index_ptr);
		/* 0. Calculate the element metrics */
		/* 0.0. Get dxi/dx and det(dxi/dx) */
		GetElemInvJ3DKernel(batch_size, ien, batch_index_ptr, xg, elem_invJ);
		/* 0.1. Calculate the gradient of the shape functions */
		for(u32 e = 0; e < batch_size; ++e) {
			A_batch[e] = elem_invJ + batch_index_ptr[e] * 10;
			B_batch[e] = shlgradu;
			C_batch[e] = shgradg + e * 12;
		}
		/* C_batch[:, :, e] = A_batch[0:3, 0:3, e].T @ B_batch[0:3, 0:NSHL, e] for e in range(new_batch_seze) */
		hipblasDgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 3, NSHL, 3, &one, A_batch, 3, B_batch, 3, &zero, C_batch, 3, batch_size);

		/* 1. Interpolate the field values */  
		hipMemset(qr_wgalpha, 0, max_batch_size * sizeof(f64));
		hipMemset(qr_dwgalpha, 0, max_batch_size * sizeof(f64));
		hipMemset(qr_wggradalpha, 0, max_batch_size * sizeof(f64) * 3);
	
		/* 1.0. Calculate the field value on the vertices */
		LoadElementValueKernel<<<num_block, num_thread>>>(batch_size, ien, batch_index_ptr, wgold_dptr, buffer);
		LoadElementValueAXPYKernel<<<num_block, num_thread>>>(batch_size, ien, batch_index_ptr, dwgold_dptr, buffer, kDT * kALPHAF * (1.0 - kGAMMA));
		LoadElementValueAXPYKernel<<<num_block, num_thread>>>(batch_size, ien, batch_index_ptr, dwg_dptr, buffer, kDT * kALPHAF * kGAMMA);

		/* 1.1. Calculate the gradient*/
		for(u32 e = 0; e < batch_size; ++e) {
			A_batch[e] = shgradg + e * 12;
			B_batch[e] = buffer + e * NSHL;
			C_batch[e] = qr_wggradalpha + e * 3;
		}
		/* C_batch[0:3, e] = A_batch[0:3, 0:NSHL, i] @ B_batch[0:NSHL, e] for e in range(batch_size) */	
		hipblasDgemvBatched(handle, HIPBLAS_OP_N, 3, NSHL, &one, A_batch, 3, B_batch, 1, &zero, C_batch, 1, batch_size);

		/* 1.2. Calculate the field value on the quadrature */
		/* qr_wgalpha[0:NQR, :] = shlu[0:NQR, 0:NSHL] @ buffer[0:NSHL, :] */
		hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, NQR, batch_size, NSHL, &one, shlu, NQR, buffer, NSHL, &zero, qr_wgalpha, NQR);

		/* 1.3. Calculate the field value of qr_dwgalpha */
		hipMemset(qr_dwgalpha, 0, num_tet * sizeof(f64) * NQR);
		LoadElementValueAXPYKernel<<<num_block, num_thread>>>(batch_size, ien, batch_index_ptr, dwgold_dptr, buffer, 1.0 - kALPHAM);
		LoadElementValueAXPYKernel<<<num_block, num_thread>>>(batch_size, ien, batch_index_ptr, dwg_dptr, buffer, kALPHAM);
		hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, NQR, batch_size, NSHL, &one, shlu, NQR, buffer, NSHL, &zero, qr_dwgalpha, NQR);


		/* 2. Calculate the elementwise residual and jacobian */
		AssemleTetWeakFormKernel<<<num_block, num_thread>>>(batch_size, batch_index_ptr, elem_invJ, qr_wgalpha, qr_dwgalpha, qr_wggradalpha, shgradg, elem_F, elem_J);
		/* 3. Assemble the global residual vector */
		if(F) {
			ElemRHSLocal2GlobalKernel<<<num_block, num_thread>>>(batch_size, batch_index_ptr, ien, elem_F, F);
		}
		/* 4. Assemble the global residual matrix */
		if(J) {
		}
	}
	CdamFreeDevice(buffer, num_tet * sizeof(f64));
	CdamFreeDevice(elem_invJ, num_tet * sizeof(f64) * 10);
	CdamFreeDevice(shgradg, num_tet * sizeof(f64) * NSHL * 3);

	CdamFreeDevice(elem_F, num_tet * sizeof(f64) * NSHL);
	CdamFreeDevice(elem_J, num_tet * sizeof(f64) * NSHL * NSHL);

	CdamFreeDevice(qr_wgalpha, num_tet * sizeof(f64));
	CdamFreeDevice(qr_dwgalpha, num_tet * sizeof(f64));
	CdamFreeDevice(qr_wggradalpha, num_tet * sizeof(f64) * 3);
	CdamFreeDevice(batch_index_ptr, batch_size * sizeof(u32));
	hipblasDestroy(handle);
}

__END_DECLS__
