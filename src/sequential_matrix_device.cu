#include "hip/hip_runtime.h"
#include "blas.h"
#include "sequential_matrix.h"

#ifdef CDAM_USE_CUDA


__BEGIN_DECLS__
__global__ void SeqMatZeroRowSparseKernel(value_type* data, index_type nrow, index_type ncol,
																					index_type nr, index_type* irow,
																					index_type rshift, index_type cshift,
																					value_type diag) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i > ncol * nr) return;
	int ir = irow[i / ncol] + rshift;
	int ic = i % ncol + cshift;
	if (ir < 0 || ir >= nrow || ic < 0 || ic >= ncol) return;

	data[ir * ncol + ic] = (ir == ic) * diag; 
}
void SeqMatZeroRowDenseGPU(value_type* data, index_type nrow, index_type ncol,
													 index_type nr, index_type* row,
													 index_type rshift, index_type cshift,
													 value_type diag, hipStream_t stream) {
	int num_threads = 256;
	int num_blocks = CEIL_DIV(n * ncol, num_threads);
	if(nrow != ncol)
		diag = 0.0;
	SeqMatZeroRowDenseKernel<<<num_blocks, num_threads, 0, stream>>>(data, nrow, ncol, n, row, rshift, cshift, diag);
}
__global__ void SeqMatGetSubmatDenseColMajorKernel(value_type* src, index_type nrow, index_type ncol,
																									 value_type* dst, index_type nr, index_type nc,
																									 index_type* row, index_type* col) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nr * nc) return;
	int ir = row[i % nr];
	int ic = col[i / nr];
	if (ir < 0 || ir >= nrow || ic < 0 || ic >= ncol) return;

	dst[i] = src[ir + ic * nrow];
}
__global__ void SeqMatGetSubmatDenseRowMajorKernel(value_type* src, index_type nrow, index_type ncol,
																									 value_type* dst, index_type nr, index_type nc,
																									 index_type* row, index_type* col) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= nr * nc) return;
	int ir = row[i / nc];
	int ic = col[i % nc];
	if (ir < 0 || ir >= nrow || ic < 0 || ic >= ncol) return;

	dst[i] = src[ir * ncol + ic];
}

void SeqMatGetSubmatDenseGPU(MatOrder order, value_type* src, index_type nrow, index_type ncol,
														 value_type* dst, index_type nr, index_type nc,
														 index_type* row, index_type* col, hipStream_t stream) {
	int num_threads = 256;
	int num_blocks = CEIL_DIV(nr * nc, num_threads);
	if (order == ROW_MAJOR) {
		SeqMatGetSubmatDenseRowMajorKernel<<<num_blocks, num_threads, 0, stream>>>(src, nrow, ncol, dst, nr, nc, row, col);
	} else {
		SeqMatGetSubmatDenseColMajorKernel<<<num_blocks, num_threads, 0, stream>>>(src, nrow, ncol, dst, nr, nc, row, col);
	}
}

__global__ void SeqMatGetDiagDenseKernel(value_type* data, index_type n, value_type* diag, index_type bs) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= k * bs * bs) return;

	int ir = i / bs;
	int ic = i % bs;

	diag[i] = data[ir * n + (ir / bs) * bs + ic]; 
}

void SeqMatGetDiagDenseGPU(value_type* data, index_type n,
													 value_type* diag, index_type bs, hipStream_t stream) {
	if(bs == 1) {
		dcopy(n, data, n + 1, diag, 1);
	}
	else (bs > 1) {
		int num_threads = 256;
		int num_blocks = CEIL_DIV(n * bs, num_threads);
		SeqMatGetDiagDenseKernel<<<num_blocks, num_threads, 0, stream>>>(data, n, diag, bs);
	}
}

__global__ void SeqMatAddElemValueBatchedDenseKernel(value_type* data, index_type nrow, index_type ncol,
																										 MatOrder order,
																										 MatStorageMethod rmap_storage, MatStorageMethod cmap_storage,
																										 index_type batch_size, index_type* batch_index_ptr,
																										 index_type* ien, index_type nshl,
																										 index_type block_row, index_type block_col,
																										 value_type* value, index_type ldv, index_type stride) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	index_type i, ir, ic, iel;
	index_type batch, ishl, jshl; ishg, jshg;
	index_type dst_row, dst_col;

	if (idx >= batch_size * nshl * nshl) return;

	iel = idx / (nshl * nshl);
	iel = batch_index_ptr[iel];
	ishl = (idx % (nshl * nshl)) / nshl;
	jshl = idx % nshl;

	ishg = ien[iel * nshl + ishl];
	jshg = ien[iel * nshl + jshl];

	value += idx * stride;
	for(ir = 0; i < block_row; i++) {
		dst_row = DOFMap(rmap_storage, nrow / block_row, block_row, ishg, ir);
		for(ic = 0; ic < block_col; ++ic) {
			dst_col = DOFMap(cmap_storage, ncol / block_col, block_col, jshg, ic);
			data[DOFMap(order, nrow, ncol, dst_row, dst_col)] += value[ir * ldv + ic];
		}
	}
}

void SeqMatAddElemValueBatchedDenseGPU(value_type* data, index_type nrow, index_type ncol,
																			 MatOrder order,
																			 MatStorageMethod rmap_storage, MatStorageMethod cmap_storage,
																			 index_type batch_size, index_type* batch_index_ptr,
																			 index_type* ien, index_type nshl,
																			 index_type block_row, index_type block_col,
																			 value_type* value, index_type ldv, index_type stride,
																			 hipStream_t stream) {
	int num_threads = 256;
	int num_blocks = CEIL_DIV(batch_size * nshl * nshl, num_threads);
	SeqMatAddElemValueBatchedDenseKernel<<<num_blocks, num_threads, 0, stream>>>(data, nrow, ncol, order, rmap_storage, cmap_storage, batch_size, batch_index_ptr, ien, nshl, block_row, block_col, value, ldv, stride);
}

__global__ void SeqMatCopySubmatValueCSRKernel(value_type* src, index_type* row_ptr, index_type* col_ind,
																							 index_type nr, index_type* row, index_type nc, index_type* col,
																							 value_type* dst, index_type* dst_row_ptr) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= nr) return;
	int j, k, ir, ic;

	index_type start = row_ptr[row[i]];
	index_type end = row_ptr[row[i] + 1];

	dst += dst_row_ptr[i];

	for(j = start; j < end; ++j) {
		for(k = 0; k < nc; ++k) {
			if(col_ind[j] == col[k]) {
				dst[k] = src[j];
			}
		}
	}
}

void SeqMatCopySubmatValueCSRGPU(value_type* src, CSRAttr* src_spy,
																 index_type nr, index_type* row, index_type nc, index_type* col,
																 value_type* dst, CSRAttr* dst_spy, hipStream_t stream) {
	int num_threads = 256;
	int num_blocks = CEIL_DIV(nr, num_threads);
	index_type* row_ptr = CSRAttrRowPtr(src_spy);
	index_type* col_ind = CSRAttrColInd(src_spy);
	index_type* dst_row_ptr = CSRAttrRowPtr(dst_spy);
	SeqMatCopySubmatValueCSRKernel<<<num_blocks, num_threads, 0, stream>>>(src, row_ptr, col_ind, nr, row, nc, col, dst, dst_row_ptr);
}

__global__ void SeqMatGetDiagBlockedCSRKernel(value_type* src, index_type nrow, index_type ncol,
																							index_type* row_ptr, index_type* col_ind,
																							index_type* diag, index_type ld, index_type stride, index_type bs) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= nrow) return;
	int j, k, start, end, ir, ic;
	int ib, lane, range[2];
	start = row_ptr[i];
	end = row_ptr[i + 1];
	ib = i / bs;
	lane = i % bs;
	range[0] = ib * bs;
	range[1] = MIN(range[0] + bs, ncol);

	src += start;
	diag += ib * stride + lane * ld; 

	for(j = start; j < end; ++j) {
		ic = col_ind[j];
		if(ic >= range[0] && ic < range[1]) {
			diag[ic - range[0]] = src[j];
		}
	}
}

void SeqMatGetDiagBlockedCSRGPU(value_type* src, CSRAttr* spy, 
																index_type* diag, index_type ld, index_type stride, index_type bs,
																hipStream_t stream) {
	int num_threads = 256;
	int num_blocks = CEIL_DIV(nrow, num_threads);
	SeqMatGetDiagBlockedCSRKernel<<<num_blocks, num_threads, 0, stream>>>(src, nrow, ncol, CSRAttrRowPtr(spy), CSRAttrColInd(spy), diag, ld, stride, bs);
}


__global__ void SeqMatAddElemValueCSRKernel(value_type alpha, value_type* matval, 
																						index_type batch_size, index_type* batch_index_ptr,
																						index_type* ien, index_type nshl,
																						index_type nrow, index_type ncol,
																						index_type* row_ptr, index_type* col_ind,
																						MatStorageMethod rmap_storage, MatStorageMethod cmap_storage,
																						index_type block_row, index_type block_col,
																						value_type beta, value_type* val, int lda, int stride) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i >= batch_size * nshl * nshl) return;
	index_type iel = batch_index_ptr[i / (nshl * nshl)];
	index_type ishl = (i % (nshl * nshl)) / nshl;
	index_type jshl = i % nshl;

	index_type ishg = ien[iel * nshl + ishl];
	index_type jshg = ien[iel * nshl + jshl];

	index_type start, end, len;

	index_type dst_row, dst_col;
	index_type ii, jj, k;

	val += i * stride;

	for(ii = 0; ii < block_row; ++ii) {
		dst_row = DOFMap(rmap_storage, nrow / block_row, block_row, ishg, ii);

		start = row_ptr[dst_row];
		end = row_ptr[dst_row + 1];
		len = end - start;

		for(jj = 0; jj < block_col; ++jj) {
			dst_col = DOFMap(cmap_storage, ncol / block_col, block_col, jshg, jj);

			for(k = start; k < end; ++k) {
				if(col_ind[k] == dst_col) {
					matval[k] = alpha * matval[k] + beta * val[ii * lda + jj];
				}
			}
		}
	}

}

void SeqMatAddElemValueCSRGPU(value_type alpha, value_type* matval,
															CSRAttr* spy, MatStorageMethod rmap_storage, MatStorageMethod cmap_storage,
															index_type batch_size, index_type* batch_index_ptr,
															index_type* ien, index_type nshl,
															index_type block_row, index_type block_col,
															value_type beta, value_type* val, int lda, int stride, hipStream_t stream) {
	int num_threads = 256;
	int num_blocks = CEIL_DIV(batch_size * nshl * nshl, num_threads);

	SeqMatAddElemValueCSRKernel<<<num_blocks, num_threads, 0, stream>>>(
			alpha, matval, batch_size, batch_index_ptr, ien, nshl,
			CSRAttrNumRow(spy), CSRAttrNumCol(spy), CSRAttrRowPtr(spy), CSRAttrColInd(spy),
			rmap_storage, cmap_storage, block_row, block_col, beta, val, lda, stride);
}
__END_DECLS__
#endif /* CDAM_USE_CUDA */
