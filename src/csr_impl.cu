#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include "alloc.h"
// #include "Mesh.h"
#include "csr.h"

#define MAX_ROW_LENGTH 64
template<typename Index>
__global__ void
GetFindRowKernel(Index nnz, Index num_rows,
								 const Index* __restrict__ row_ptr,
								 Index* __restrict__ find_row) {
	Index i = blockIdx.x * blockDim.x + threadIdx.x;
	while(i < num_rows) {
		for(Index j = row_ptr[i]; j < row_ptr[i + 1]; j++) {
			find_row[j] = i;
		}
		i += blockDim.x * gridDim.x;
	}
}

template<typename Index>
__global__ void
SetRowLength(Index num_rows, const Index* __restrict__ row_ptr,
						 Index block_row, Index block_col,
						 Index* __restrict__ new_row_ptr) {
	Index i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= num_rows) return;
	Index start = row_ptr[i];
	Index len = row_ptr[i + 1] - start;
	for(Index j = 0; j < block_row; j++) {
		new_row_ptr[i * block_row + j] = start * block_row * block_col + j * block_col * len;
	}
}

template<typename Index>
__global__ void
SetColIndex(Index nnz, Index num_rows, 
						const Index* __restrict__ row_ptr, const Index* __restrict__ col_ind,
						Index block_row, Index block_col,
						const Index* __restrict__ new_row_ptr, Index* __restrict__ new_col_ind) {

	Index i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= num_rows) return;
	Index start = row_ptr[i], end = row_ptr[i + 1];
	Index len = end - start;

	for (Index j = 0; j < block_row; j++) {
		Index row = i * block_row + j;
		for(Index k = 0; k < len; k++) {
			Index col = col_ind[start + k];
			for(Index l = 0; l < block_col; l++) {
				new_col_ind[new_row_ptr[row] + k * block_col + l] = col * block_col + l;
			}
		}
	}
}

/*
template<typename Index, int NSHL=4>
__global__ void
GenerateV2VMap(Index num_elem, const Index* __restrict__ ien, Index* __restrict__ v2e_map) {
	Index b_start = blockIdx.x * blockDim.x;
	Index b_end = min(b_start + blockDim.x, num_elem);
	Index i = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ Index local_ien[NSHL * blockDim.x];
	__shared__ Index local_v2v_map[MAX_BATCH_LENGTH * blockDim.x];
	while(i < num_elem) {
		for(Index j = 0; j < NSHL; ++j) {
			local_ien[threadIdx.x + j * (b_end - b_start)] = ien[i + j * (b_end - b_start)];
		}
		for(Index j = 0; j < MAX_BATCH_LENGTH; ++j) {
			local_v2v_map[threadIdx.x + j * blockDim.x] = 0;
		}
		__syncthreads();
		

		v2e_map[i * MAX_BATCH_LENGTH] = 0;
		for(Index j = 0; j < NSHL; j++) {
			Index node = elem[j];
			Index c = atomicAdd(v2e_map + node * MAX_BATCH_LENGTH, 1);
			v2e_map[node * MAX_BATCH_LENGTH + c + 1] = i;
		}
		i += blockDim.x * gridDim.x;
	}
}
*/

template <typename I>
__global__ void
CSRAttrGetNZIndBatchedKernel(I num_row, I num_col, const I* row_ptr, const I* col_ind,
														 I batch_size, const I* row, const I* col, I* ind) {
	I i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i >= batch_size) return;

	I row_start = row_ptr[row[i]];
	I row_end = row_ptr[row[i] + 1];

	for(I j = row_start; j < row_end; j++) {
		if(col[j] == col[i]) {
			ind[i] = j;
			return;
		}
	}
}


__BEGIN_DECLS__

typedef csr_index_type index_t;
// void GenerateCSRFromMesh(const Mesh3D* mesh, CSRAttr* attr) {
// 	index_t num_node = Mesh3DNumNode(mesh);
// 	index_t num_elem = Mesh3DNumTet(mesh);
// 	const Mesh3DData* device = Mesh3DDevice(mesh);
// 	index_type* ien = device->ien;
// 	index_t* buffer = (index_t*)CdamMallocDevice(SIZE_OF(index_t) * num_node * MAX_ROW_LENGTH);
// 	/* Generate a vertex-to-vetex mapping */
// 	hipMemset(buffer, 0, SIZE_OF(index_t) * num_node * MAX_ROW_LENGTH);
// 	fprintf(stderr, "Not implemented\n");
// 
// 	CdamFreeDevice(buffer, SIZE_OF(index_t) * num_node * MAX_ROW_LENGTH);
// }

void ExpandCSRByBlockSize(const CSRAttr* attr, CSRAttr* new_attr, index_t block_size[2]) {
	index_t num_rows = CSRAttrNumRow(attr);
	index_t num_cols = CSRAttrNumCol(attr);
	index_t nnz = CSRAttrNNZ(attr);
	index_t* row_ptr = CSRAttrRowPtr(attr);
	index_t* col_ind = CSRAttrColInd(attr);

	index_t block_row = block_size[0];
	index_t block_col = block_size[1];
	hipStream_t stream;
	hipStreamCreate(&stream);

	CSRAttrNumRow(new_attr) = num_rows * block_row;
	CSRAttrNumCol(new_attr) = num_cols * block_col;
	CSRAttrNNZ(new_attr) = nnz * block_row * block_col;

	// CSRAttrRowPtr(new_attr) = (index_t*)CdamMallocDevice(SIZE_OF(index_t) * (CSRAttrNumRow(new_attr) + 1));
	// CSRAttrColInd(new_attr) = (index_t*)CdamMallocDevice(SIZE_OF(index_t) * CSRAttrNNZ(new_attr));
	// index_t* find_row = (index_t*)CdamMallocDevice(SIZE_OF(index_t) * nnz);

	int block_dim = 256;
	int block_num = (num_rows + block_dim - 1) / block_dim;
	// GetFindRowKernel<<<block_num, block_dim, 0, stream>>>(nnz, num_rows, row_ptr, find_row);
	SetRowLength<<<CEIL_DIV(num_rows, block_dim), block_dim, 0, stream>>>(num_rows, row_ptr, block_row, block_col, CSRAttrRowPtr(new_attr));
	SetColIndex<<<CEIL_DIV(num_rows, block_dim), block_dim, 0, stream>>>(nnz, num_rows, row_ptr, col_ind, block_row, block_col, CSRAttrRowPtr(new_attr), CSRAttrColInd(new_attr));

	// CdamFreeDevice(find_row, SIZE_OF(index_t) * nnz);
	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);

}

void CSRAttrGetNZIndBatchedGPU(const CSRAttr* attr,
																 csr_index_type batch_size, const csr_index_type* row, const csr_index_type* col,
																 csr_index_type* ind) {
	csr_index_type num_rows = CSRAttrNumRow(attr);
	csr_index_type num_cols = CSRAttrNumCol(attr);
	csr_index_type nnz = CSRAttrNNZ(attr);
	const csr_index_type* row_ptr = CSRAttrRowPtr(attr);
	const csr_index_type* col_ind = CSRAttrColInd(attr);

	int block_dim = 256;
	int grid_dim = (batch_size + block_dim - 1) / block_dim;
	CSRAttrGetNZIndBatchedKernel<index_type><<<grid_dim, block_dim>>>(num_rows, num_cols, row_ptr, col_ind, /* csr */
																																	  batch_size, row, col, ind);

}

__END_DECLS__
